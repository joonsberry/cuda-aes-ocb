/* Project: CUDA AES-OCB
 * Author: Jonathan Kenney
 *
 * Much of this source is thanks to Github user DesWurstes
 * See their work on pure C AES: https://github.com/DesWurstes/OCB-AES
 * All host functions and constants were contrived by DesWurstes,
 * the goal of this module is to achieve CUDA implementation for the
 * full cipher block computations.
 *
 * Original acknowledgements from DesWurstes:
 *
 * AES OCB Licenses: http://web.cs.ucdavis.edu/~rogaway/ocb/license.htm
 * Applied license: License 1
 * Found at: http://web.cs.ucdavis.edu/~rogaway/ocb/license1.pdf
 * OCB FAQ: http://www.cs.ucdavis.edu/~rogaway/ocb/ocb-faq.htm
 *
 * The source code is derived from this, except the blockcipher functions:
 * https://tools.ietf.org/pdf/rfc7253.pdf
 *
 * For the curious:
 * Unneeded extra: https://csrc.nist.gov/csrc/media/publications/fips/197/final/documents/fips-197.pdf
 * More extra: https://link.springer.com/content/pdf/10.1007%2F978-3-642-21702-9_18.pdf
 *
 * Cipher functions are taken from (Public domain)
 * https://github.com/kokke/tiny-AES-c/blob/master/aes.c
*/


#include <hip/hip_runtime.h>
#include <time.h>

#ifdef __GNUC__
#define USE_BUILTIN
#define ocb_ntz(a) __builtin_ctz((unsigned int) a)
#define ocb_ntz_round(a) \
((a) == 0) ? 0 : (sizeof(unsigned int) * 8 - __builtin_clz((unsigned int) (a)) - 1)
#define ocb_memcpy(a,b,c) __builtin_memcpy(a,b,c)
#else
#define ocb_memcpy(a,b,c) \
for (int _i = 0; _i < (c); _i++) \
  a[_i] = b[_i];
#endif

#ifdef OCB_CONST_NONCE
#define OCB_NONCEPARAM
#define nonce_length OCB_CONST_NONCE
#else
#define OCB_NONCEPARAM unsigned int nonce_length,
#endif

//--------------------HOST CONSTANTS--------------------

static const unsigned char sbox[256] = {
  //0     1     2     3     4     5     6     7     8     9     A     B     C     D     E     F
  0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
  0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
  0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
  0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
  0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
  0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
  0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
  0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
  0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
  0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
  0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
  0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
  0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
  0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
  0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
  0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16};

static const unsigned char rsbox[256] = {
  0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
  0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
  0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
  0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
  0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
  0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
  0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
  0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
  0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
  0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
  0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
  0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
  0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
  0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
  0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
  0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d};

  static const unsigned char rcon[11] = {0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36};

//--------------------CUDA CONSTANTS--------------------

__constant__ unsigned char *d_sbox;
__constant__ unsigned char *d_rsbox;

//--------------------HOST FUNCTIONS--------------------

// The SubBytes Function Substitutes the values in the
// state matrix with values in an S-box.
static void sub_bytes(unsigned char state[16])
{
  for (int i = 0; i < 16; i++)
    state[i] = sbox[state[i]];
}

static void inv_sub_bytes(unsigned char state[16])
{
  for (int i = 0; i < 16; i++)
    state[i] = rsbox[state[i]];
}

// The ShiftRows() function shifts the rows in the state to the left.
// Each row is shifted with different offset.
// Offset = Row number. So the first row is not shifted.
static void shift_rows(unsigned char state[16])
{
  unsigned char temp;

  // Rotate first row 1 columns to left
  temp        = state[0 * 4 + 1];
  state[0 * 4 + 1] = state[1 * 4 + 1];
  state[1 * 4 + 1] = state[2 * 4 + 1];
  state[2 * 4 + 1] = state[3 * 4 + 1];
  state[3 * 4 + 1] = temp;

  // Rotate second row 2 columns to left
  temp        = state[0 * 4 + 2];
  state[0 * 4 + 2] = state[2 * 4 + 2];
  state[2 * 4 + 2] = temp;

  temp        = state[1 * 4 + 2];
  state[1 * 4 + 2] = state[3 * 4 + 2];
  state[3 * 4 + 2] = temp;

  // Rotate third row 3 columns to left
  temp        = state[0 * 4 + 3];
  state[0 * 4 + 3] = state[3 * 4 + 3];
  state[3 * 4 + 3] = state[2 * 4 + 3];
  state[2 * 4 + 3] = state[1 * 4 + 3];
  state[1 * 4 + 3] = temp;
}

static void inv_shift_rows(unsigned char state[16])
{
  unsigned char temp;

  // Rotate first row 1 columns to right
  temp = state[3 * 4 + 1];
  state[3 * 4 + 1] = state[2 * 4 + 1];
  state[2 * 4 + 1] = state[1 * 4 + 1];
  state[1 * 4 + 1] = state[0 * 4 + 1];
  state[0 * 4 + 1] = temp;

  // Rotate second row 2 columns to right
  temp = state[0 * 4 + 2];
  state[0 * 4 + 2] = state[2 * 4 + 2];
  state[2 * 4 + 2] = temp;

  temp = state[1 * 4 + 2];
  state[1 * 4 + 2] = state[3 * 4 + 2];
  state[3 * 4 + 2] = temp;

  // Rotate third row 3 columns to right
  temp = state[0 * 4 + 3];
  state[0 * 4 + 3] = state[1 * 4 + 3];
  state[1 * 4 + 3] = state[2 * 4 + 3];
  state[2 * 4 + 3] = state[3 * 4 + 3];
  state[3 * 4 + 3] = temp;
}

// This function adds the round key to state.
// The round key is added to the state by an XOR function.
static void add_round_key(unsigned char round, unsigned char state[16], const unsigned char * __restrict round_key)
{
  for (int i = 0; i < 16; ++i)
    state[i] ^= round_key[(round * 16) + i];
}

static inline unsigned char xtime(unsigned char x)
{
  return ((x << 1) ^ (((x >> 7) & 1) * 0x1b));
}

static inline unsigned char Multiply(unsigned char x, unsigned char y) {
  return (((y & 1) * x) ^
    ((y >> 1 & 1) * xtime(x)) ^
    ((y >> 2 & 1) * xtime(xtime(x))) ^
    ((y >> 3 & 1) * xtime(xtime(xtime(x)))));
}

// MixColumns function mixes the columns of the state matrix
static void mix_columns(unsigned char state[16])
{
  unsigned char Tmp, Tm, t;
  for (int i = 0; i < 4; i++)
  {
    t   = state[4 * i + 0];
    Tmp = state[4 * i + 0] ^ state[4 * i + 1] ^ state[4 * i + 2] ^ state[4 * i + 3];
    Tm  = state[4 * i + 0] ^ state[4 * i + 1] ; Tm = xtime(Tm);  state[4 * i + 0] ^= Tm ^ Tmp;
    Tm  = state[4 * i + 1] ^ state[4 * i + 2] ; Tm = xtime(Tm);  state[4 * i + 1] ^= Tm ^ Tmp;
    Tm  = state[4 * i + 2] ^ state[4 * i + 3] ; Tm = xtime(Tm);  state[4 * i + 2] ^= Tm ^ Tmp;
    Tm  = state[4 * i + 3] ^ t ;           Tm = xtime(Tm);  state[4 * i + 3] ^= Tm ^ Tmp;
  }
}

static void inv_mix_columns(unsigned char state[16])
{
  unsigned char a, b, c, d;
  for (int i = 0; i < 4; i++)
  {
    a = state[4 * i + 0];
    b = state[4 * i + 1];
    c = state[4 * i + 2];
    d = state[4 * i + 3];

    state[4 * i + 0] = Multiply(a, 0x0e) ^ Multiply(b, 0x0b) ^ Multiply(c, 0x0d) ^ Multiply(d, 0x09);
    state[4 * i + 1] = Multiply(a, 0x09) ^ Multiply(b, 0x0e) ^ Multiply(c, 0x0b) ^ Multiply(d, 0x0d);
    state[4 * i + 2] = Multiply(a, 0x0d) ^ Multiply(b, 0x09) ^ Multiply(c, 0x0e) ^ Multiply(d, 0x0b);
    state[4 * i + 3] = Multiply(a, 0x0b) ^ Multiply(b, 0x0d) ^ Multiply(c, 0x09) ^ Multiply(d, 0x0e);
  }
}

// Cipher is the main function that encrypts the PlainText.
// round_key is of len 240 chars.
static void cipher(unsigned char state[16], const unsigned char * __restrict round_key)
{
  // Add the First round key to the state before starting the rounds.
  add_round_key(0, state, round_key);

  // There will be Nr rounds.
  // The first Nr-1 rounds are identical.
  // These Nr-1 rounds are executed in the loop below.
  for (unsigned char round = 1; round < 14; round++)
  {
    sub_bytes(state);
    shift_rows(state);
    mix_columns(state);
    add_round_key(round, state, round_key);
  }

  // The last round is given below.
  // The MixColumns function is not here in the last round.
  sub_bytes(state);
  shift_rows(state);
  add_round_key(14, state, round_key);
}

static void decipher(unsigned char state[16], const unsigned char * __restrict round_key)
{

  // Add the First round key to the state before starting the rounds.
  add_round_key(14, state, round_key);

  // There will be Nr rounds.
  // The first Nr-1 rounds are identical.
  // These Nr-1 rounds are executed in the loop below.
  for (unsigned char round = 13; round > 0; round--)
  {
    inv_shift_rows(state);
    inv_sub_bytes(state);
    add_round_key(round, state, round_key);
    inv_mix_columns(state);
  }

  // The last round is given below.
  // The MixColumns function is not here in the last round.
  inv_shift_rows(state);
  inv_sub_bytes(state);
  add_round_key(0, state, round_key);
}

static void key_expansion(unsigned char * __restrict round_key, const unsigned char * __restrict key)
{
  unsigned i, j, k;
  unsigned char tempa[4]; // Used for the column/row operations

  // The first round key is the key itself.
  ocb_memcpy(round_key, key, 32);

  // All other round keys are found from the previous round keys.
  for (i = 8; i < 4 * (14 + 1); ++i)
  {
    {
      k = (i - 1) * 4;
      tempa[0] = round_key[k + 0];
      tempa[1] = round_key[k + 1];
      tempa[2] = round_key[k + 2];
      tempa[3] = round_key[k + 3];
    }

    if (i % 8 == 0)
    {
      // This function shifts the 4 bytes in a word to the left once.
      // [a0,a1,a2,a3] becomes [a1,a2,a3,a0]

      // Function RotWord()
      {
        k = tempa[0];
        tempa[0] = tempa[1];
        tempa[1] = tempa[2];
        tempa[2] = tempa[3];
        tempa[3] = k;
      }

      // SubWord() is a function that takes a four-byte input word and
      // applies the S-box to each of the four bytes to produce an output word.

      // Function Subword()
      {
        tempa[0] = sbox[tempa[0]];
        tempa[1] = sbox[tempa[1]];
        tempa[2] = sbox[tempa[2]];
        tempa[3] = sbox[tempa[3]];
      }

      tempa[0] = tempa[0] ^ rcon[i/8];
    }
    if (i % 8 == 4)
    {
      // Function Subword()
      {
        tempa[0] = sbox[tempa[0]];
        tempa[1] = sbox[tempa[1]];
        tempa[2] = sbox[tempa[2]];
        tempa[3] = sbox[tempa[3]];
      }
    }
    j = i * 4; k = (i - 8) * 4;
    round_key[j + 0] = round_key[k + 0];
    round_key[j + 1] = round_key[k + 1];
    round_key[j + 2] = round_key[k + 2];
    round_key[j + 3] = round_key[k + 3];
    round_key[j + 0] ^= tempa[0];
    round_key[j + 1] ^= tempa[1];
    round_key[j + 2] ^= tempa[2];
    round_key[j + 3] ^= tempa[3];
  }
}
// End of AES common bundle


static void double_arr(unsigned char s[16]) {
  const unsigned char first_bit = -(s[0] >> 7);
  for (int i = 0; i < 15; i++) {
    s[i] &= 127;
    s[i] <<= 1;
    s[i] |= s[i+1] >> 7;
  }
  s[15] &= 127;
  s[15] <<= 1;
  s[15] ^= first_bit & 135;
}

#ifndef USE_BUILTIN
// largest x such that 2^x | a - n for a - n > 0
static inline unsigned int ocb_ntz_round(unsigned int a) {
  int k = 0;
  while (a >>= 1)
    k++;
  return (unsigned int) k;
}

// largest x such that 2^x | a
static inline unsigned int ocb_ntz(unsigned int a) {
  int k = 0;
  while ((a % 2 == 0) && (a >>= 1))
    k++;
  return (unsigned int) k;
}
#endif

static inline void xor_16(unsigned char * __restrict a, const unsigned char * __restrict b) {
  for (int i = 0; i < 16; i++)
    a[i] ^= b[i];
}

//--------------------CUDA KERNELS/FUNCTIONS--------------------

// The SubBytes Function Substitutes the values in the
// state matrix with values in an S-box.
__device__ void cuda_sub_bytes(unsigned char state[16])
{
  for (int i = 0; i < 16; i++)
    state[i] = d_sbox[state[i]];
}

__device__ void cuda_inv_sub_bytes(unsigned char state[16])
{
  for (int i = 0; i < 16; i++)
    state[i] = d_rsbox[state[i]];
}

// The ShiftRows() function shifts the rows in the state to the left.
// Each row is shifted with different offset.
// Offset = Row number. So the first row is not shifted.
__device__ void cuda_shift_rows(unsigned char state[16])
{
  unsigned char temp;

  // Rotate first row 1 columns to left
  temp        = state[0 * 4 + 1];
  state[0 * 4 + 1] = state[1 * 4 + 1];
  state[1 * 4 + 1] = state[2 * 4 + 1];
  state[2 * 4 + 1] = state[3 * 4 + 1];
  state[3 * 4 + 1] = temp;

  // Rotate second row 2 columns to left
  temp        = state[0 * 4 + 2];
  state[0 * 4 + 2] = state[2 * 4 + 2];
  state[2 * 4 + 2] = temp;

  temp        = state[1 * 4 + 2];
  state[1 * 4 + 2] = state[3 * 4 + 2];
  state[3 * 4 + 2] = temp;

  // Rotate third row 3 columns to left
  temp        = state[0 * 4 + 3];
  state[0 * 4 + 3] = state[3 * 4 + 3];
  state[3 * 4 + 3] = state[2 * 4 + 3];
  state[2 * 4 + 3] = state[1 * 4 + 3];
  state[1 * 4 + 3] = temp;
}

__device__ void cuda_inv_shift_rows(unsigned char state[16])
{
  unsigned char temp;

  // Rotate first row 1 columns to right
  temp = state[3 * 4 + 1];
  state[3 * 4 + 1] = state[2 * 4 + 1];
  state[2 * 4 + 1] = state[1 * 4 + 1];
  state[1 * 4 + 1] = state[0 * 4 + 1];
  state[0 * 4 + 1] = temp;

  // Rotate second row 2 columns to right
  temp = state[0 * 4 + 2];
  state[0 * 4 + 2] = state[2 * 4 + 2];
  state[2 * 4 + 2] = temp;

  temp = state[1 * 4 + 2];
  state[1 * 4 + 2] = state[3 * 4 + 2];
  state[3 * 4 + 2] = temp;

  // Rotate third row 3 columns to right
  temp = state[0 * 4 + 3];
  state[0 * 4 + 3] = state[1 * 4 + 3];
  state[1 * 4 + 3] = state[2 * 4 + 3];
  state[2 * 4 + 3] = state[3 * 4 + 3];
  state[3 * 4 + 3] = temp;
}

// This function adds the round key to state.
// The round key is added to the state by an XOR function.
__device__ void cuda_add_round_key(unsigned char round, unsigned char state[16], const unsigned char * __restrict round_key)
{
  for (int i = 0; i < 16; ++i)
    state[i] ^= round_key[(round * 16) + i];
}

__device__ unsigned char cuda_xtime(unsigned char x)
{
  return ((x << 1) ^ (((x >> 7) & 1) * 0x1b));
}

__device__ unsigned char cuda_Multiply(unsigned char x, unsigned char y) {
  return (((y & 1) * x) ^
    ((y >> 1 & 1) * cuda_xtime(x)) ^
    ((y >> 2 & 1) * cuda_xtime(cuda_xtime(x))) ^
    ((y >> 3 & 1) * cuda_xtime(cuda_xtime(cuda_xtime(x)))));
}

// MixColumns function mixes the columns of the state matrix
__device__ void cuda_mix_columns(unsigned char state[16])
{
  unsigned char Tmp, Tm, t;
  for (int i = 0; i < 4; i++)
  {
    t   = state[4 * i + 0];
    Tmp = state[4 * i + 0] ^ state[4 * i + 1] ^ state[4 * i + 2] ^ state[4 * i + 3];
    Tm  = state[4 * i + 0] ^ state[4 * i + 1] ; Tm = cuda_xtime(Tm);  state[4 * i + 0] ^= Tm ^ Tmp;
    Tm  = state[4 * i + 1] ^ state[4 * i + 2] ; Tm = cuda_xtime(Tm);  state[4 * i + 1] ^= Tm ^ Tmp;
    Tm  = state[4 * i + 2] ^ state[4 * i + 3] ; Tm = cuda_xtime(Tm);  state[4 * i + 2] ^= Tm ^ Tmp;
    Tm  = state[4 * i + 3] ^ t ;           Tm = cuda_xtime(Tm);  state[4 * i + 3] ^= Tm ^ Tmp;
  }
}

__device__ void cuda_inv_mix_columns(unsigned char state[16])
{
  unsigned char a, b, c, d;
  for (int i = 0; i < 4; i++)
  {
    a = state[4 * i + 0];
    b = state[4 * i + 1];
    c = state[4 * i + 2];
    d = state[4 * i + 3];

    state[4 * i + 0] = cuda_Multiply(a, 0x0e) ^ cuda_Multiply(b, 0x0b) ^ cuda_Multiply(c, 0x0d) ^ cuda_Multiply(d, 0x09);
    state[4 * i + 1] = cuda_Multiply(a, 0x09) ^ cuda_Multiply(b, 0x0e) ^ cuda_Multiply(c, 0x0b) ^ cuda_Multiply(d, 0x0d);
    state[4 * i + 2] = cuda_Multiply(a, 0x0d) ^ cuda_Multiply(b, 0x09) ^ cuda_Multiply(c, 0x0e) ^ cuda_Multiply(d, 0x0b);
    state[4 * i + 3] = cuda_Multiply(a, 0x0b) ^ cuda_Multiply(b, 0x0d) ^ cuda_Multiply(c, 0x09) ^ cuda_Multiply(d, 0x0e);
  }
}

// Cipher is the main function that encrypts the PlainText.
// round_key is of len 240 chars.
__device__ void cuda_cipher(unsigned char state[16], const unsigned char * __restrict round_key)
{
  // Add the First round key to the state before starting the rounds.
  cuda_add_round_key(0, state, round_key);

  // There will be Nr rounds.
  // The first Nr-1 rounds are identical.
  // These Nr-1 rounds are executed in the loop below.
  for (unsigned char round = 1; round < 14; round++)
  {
    cuda_sub_bytes(state);
    cuda_shift_rows(state);
    cuda_mix_columns(state);
    cuda_add_round_key(round, state, round_key);
  }

  // The last round is given below.
  // The MixColumns function is not here in the last round.
  cuda_sub_bytes(state);
  cuda_shift_rows(state);
  cuda_add_round_key(14, state, round_key);
}

__device__ void cuda_decipher(unsigned char state[16], const unsigned char * __restrict round_key)
{

  // Add the First round key to the state before starting the rounds.
  cuda_add_round_key(14, state, round_key);

  // There will be Nr rounds.
  // The first Nr-1 rounds are identical.
  // These Nr-1 rounds are executed in the loop below.
  for (unsigned char round = 13; round > 0; round--)
  {
    cuda_inv_shift_rows(state);
    cuda_inv_sub_bytes(state);
    cuda_add_round_key(round, state, round_key);
    cuda_inv_mix_columns(state);
  }

  // The last round is given below.
  // The MixColumns function is not here in the last round.
  cuda_inv_shift_rows(state);
  cuda_inv_sub_bytes(state);
  cuda_add_round_key(0, state, round_key);
}

// largest x such that 2^x | a
__device__ unsigned int cuda_ocb_ntz(unsigned int a) {
  int k = 0;
  while ((a % 2 == 0) && (a >>= 1))
    k++;
  return (unsigned int) k;
}

__device__ void cuda_xor_16(unsigned char * __restrict a, const unsigned char * __restrict b) {
  for (int i = 0; i < 16; i++)
    a[i] ^= b[i];
}

//__global__ static void pllize(unsigned char *sbox, unsigned char *rsbox, unsigned char *l, unsigned char *offset, unsigned char *round_key, unsigned char *out) {
__global__ void penc(unsigned char **l, unsigned char **offsets, unsigned char *round_key, unsigned char *out) {
  
  cuda_xor_16(&out[blockIdx.x * 16], offsets[blockIdx.x]);
  cuda_cipher(&out[blockIdx.x * 16], round_key);
  cuda_xor_16(&out[blockIdx.x * 16], offsets[blockIdx.x]);
  
}

__global__ void pdec(unsigned char **l, unsigned char **offsets, unsigned char *round_key, unsigned char *out) {
  
  cuda_xor_16(&out[blockIdx.x * 16], offsets[blockIdx.x]);
  cuda_decipher(&out[blockIdx.x * 16], round_key);
  cuda_xor_16(&out[blockIdx.x * 16], offsets[blockIdx.x]);
  
}

static void pocb_encrypt(const unsigned char * __restrict key, const unsigned char * __restrict nonce, OCB_NONCEPARAM
  const unsigned char * __restrict message, unsigned int message_length, unsigned char * out) {
  
  time_t t;
  
  const unsigned int m = message_length / 16;
  const unsigned int l_length = ocb_ntz_round(m) + 1;

  unsigned char l[l_length][16];
  unsigned char l_asterisk[16] = {0};
  unsigned char l_dollar[16];
  unsigned char round_key[240];
  key_expansion(round_key, key);

  cipher(l_asterisk, round_key);
  // L_* ^^
  for (int i = 0; i < 16; i++)
    l[0][i] = l_asterisk[i];
  double_arr(l[0]);
  for (int i = 0; i < 16; i++)
    l_dollar[i] = l[0][i];
  double_arr(l[0]);
  // L_0 ^^^
  for (unsigned int i = 1; i < l_length; i++) {
    for (int k = 0; k < 16; k++)
      l[i][k] = l[i - 1][k];
    double_arr(l[i]);
  }
  unsigned char offset[24] = {0};
  int index = 15 - nonce_length;
  offset[index++] |= 1;
  for (unsigned int i = 0; i < nonce_length; index++, i++)
    offset[index] = nonce[i];
  unsigned int bottom = offset[15] % 64;
  offset[15] ^= bottom;
  cipher(offset, round_key);
  for (int i = 0; i < 8; i++)
    offset[16 + i] = offset[i];
  for (int i = 0; i < 8; i++)
    offset[16 + i] ^= offset[i + 1];

  const unsigned int shift = bottom / 8;
  const unsigned int bit_shift = bottom % 8;
  for (int i = 0; i < 16; i++)
    offset[i] = ((offset[i + shift] << bit_shift) | (offset[i + shift + 1] >> (8 - bit_shift))) & 255;

  ocb_memcpy(out, message, message_length);

  unsigned char offsets[m][16];
  
  for(int i = 0; i < 16; i++) {
    offsets[0][i] = offset[i];
  }

  for (int i = 0; i < m-1; i++) {
    xor_16(offsets[i], l[ocb_ntz(i + 1)]);
    memcpy(offsets[i+1], offsets[i], 16);
  }

  xor_16(offsets[m-1], l[ocb_ntz(m)]);

  unsigned char *d_sbox, *d_rsbox, **d_l, **d_offsets, *d_round_key, *d_out;

  // cudaMalloc((void**)&d_sbox, 256 * sizeof(char));
  // cudaMalloc((void**)&d_rsbox, 256 * sizeof(char));
  hipMalloc((void**)&d_l, l_length * 16 * sizeof(char));
  hipMalloc((void**)&d_offsets, m * 16 * sizeof(char));
  hipMalloc((void**)&d_round_key, 240 * sizeof(char));
  hipMalloc((void**)&d_out, message_length * sizeof(char));

  hipMemcpyToSymbol(HIP_SYMBOL(d_sbox), sbox, 256 * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_rsbox), rsbox, 256 * sizeof(char), hipMemcpyHostToDevice);
  
  hipMemcpy(d_l, l, l_length * 16 * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_offsets, offsets, m * 16 * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_round_key, round_key, 240 * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, message_length * sizeof(char), hipMemcpyHostToDevice);

  t = clock();
  penc<<<m, 1>>>(d_l, d_offsets, d_round_key, d_out);
  t = clock() - t;

  printf("Parallel Encipher Time: %d\n", t * 1000 / CLOCKS_PER_SEC);

  hipMemcpy(out, d_out, message_length * sizeof(char), hipMemcpyDeviceToHost);

  hipFree(d_sbox);
  hipFree(d_rsbox);
  hipFree(d_l);
  hipFree(d_offsets);
  hipFree(d_round_key);
  hipFree(d_out);

  for(int i = 0; i < 16; i++) {
    offset[i] = offsets[m-1][i];
  }

  const unsigned int p_asterisk_length = (unsigned int) (message_length % 16);
  const unsigned int full_block_length = message_length ^ p_asterisk_length;
  unsigned char checksum[16] = {0};

  for (unsigned int i = 0; i < full_block_length; i++)
    checksum[i % 16] ^= message[i];

  if (p_asterisk_length > 0) {
    xor_16(offset, l_asterisk);
    for (int i = 0; i < 16; i++)
      out[full_block_length + i] = offset[i];
    cipher(&out[full_block_length], round_key);
    // ^^pad
    for (unsigned int i = 0; i < p_asterisk_length; i++)
      out[full_block_length + i] ^= message[full_block_length + i];
    for (unsigned int i = 0; i < p_asterisk_length; i++)
      checksum[i] ^= message[full_block_length + i];
    checksum[p_asterisk_length] ^= 0x80;
  }
  xor_16(checksum, offset);
  xor_16(checksum, l_dollar);
  cipher(checksum, round_key);

  for (int i = 0; i < 16; i++)
    out[full_block_length + p_asterisk_length + i] = checksum[i];
}

static int pocb_decrypt(const unsigned char * __restrict key, const unsigned char * __restrict nonce, OCB_NONCEPARAM
  const unsigned char * __restrict encrypted, unsigned int encrypted_length, unsigned char * __restrict out) {
  const unsigned int m = encrypted_length / 16;
  const unsigned int l_length = ocb_ntz_round(m) + 1;
  unsigned char l[l_length][16];
  unsigned char l_asterisk[16] = {0};
  unsigned char l_dollar[16];
  unsigned char round_key[240];
  key_expansion(round_key, key);

  cipher(l_asterisk, round_key);
  // L_* ^^
  for (int i = 0; i < 16; i++)
    l[0][i] = l_asterisk[i];
  double_arr(l[0]);
  for (int i = 0; i < 16; i++)
    l_dollar[i] = l[0][i];
  double_arr(l[0]);
  // L_0 ^^^
  for (unsigned int i = 1; i < l_length; i++) {
    for (int k = 0; k < 16; k++)
      l[i][k] = l[i - 1][k];
    double_arr(l[i]);
  }
  unsigned char offset[24] = {0};
  int index = 15 - nonce_length;
  offset[index++] |= 1;
  for (unsigned int i = 0; i < nonce_length; index++, i++)
    offset[index] = nonce[i];
  unsigned int bottom = offset[15] % 64;
  offset[15] ^= bottom;
  cipher(offset, round_key);
  for (int i = 0; i < 8; i++)
    offset[16 + i] = offset[i];
  for (int i = 0; i < 8; i++)
    offset[16 + i] ^= offset[i + 1];

  const unsigned int shift = bottom / 8;
  const unsigned int bit_shift = bottom % 8;
  for (int i = 0; i < 16; i++)
    offset[i] = ((offset[i + shift] << bit_shift) | (offset[i + shift + 1] >> (8 - bit_shift))) & 255;

  const unsigned int c_asterisk_length = (unsigned int) (encrypted_length % 16);
  const unsigned int full_block_length = encrypted_length ^ c_asterisk_length;

  ocb_memcpy(out, encrypted, full_block_length);

  unsigned char offsets[m][16];

  for(int i = 0; i < 16; i++) {
    offsets[0][i] = offset[i];
  }

  for (int i = 0; i < m-1; i++) {
    xor_16(offsets[i], l[ocb_ntz(i + 1)]);
    memcpy(offsets[i+1], offsets[i], 16);
  }

  xor_16(offsets[m-1], l[ocb_ntz(m)]);

  unsigned char *d_sbox, *d_rsbox, **d_l, **d_offsets, *d_round_key, *d_out;

  // cudaMalloc((void**)&d_sbox, 256 * sizeof(char));
  // cudaMalloc((void**)&d_rsbox, 256 * sizeof(char));
  hipMalloc((void**)&d_l, l_length * 16 * sizeof(char));
  hipMalloc((void**)&d_offsets, m * 16 * sizeof(char));
  hipMalloc((void**)&d_round_key, 240 * sizeof(char));
  hipMalloc((void**)&d_out, full_block_length * sizeof(char));

  hipMemcpyToSymbol(HIP_SYMBOL(d_sbox), sbox, 256 * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_rsbox), rsbox, 256 * sizeof(char), hipMemcpyHostToDevice);
  
  hipMemcpy(d_l, l, l_length * 16 * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_offsets, offsets, m * 16 * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_round_key, round_key, 240 * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, full_block_length * sizeof(char), hipMemcpyHostToDevice);

  pdec<<<m, 1>>>(d_l, d_offsets, d_round_key, d_out);

  hipMemcpy(out, d_out, full_block_length * sizeof(char), hipMemcpyDeviceToHost);

  hipFree(d_sbox);
  hipFree(d_rsbox);
  hipFree(d_l);
  hipFree(d_offsets);
  hipFree(d_round_key);
  hipFree(d_out);

  for(int i = 0; i < 16; i++) {
    offset[i] = offsets[m-1][i];
  }

  unsigned char checksum[16] = {0};

  for (unsigned int i = 0; i < full_block_length; i++)
    checksum[i % 16] ^= out[i];

  if (c_asterisk_length > 0) {
    xor_16(offset, l_asterisk);
    unsigned char pad[16];
    for (int i = 0; i < 16; i++)
      pad[i] = offset[i];
    cipher(pad, round_key);
    // ^^pad
    for (unsigned int i = 0; i < c_asterisk_length; i++)
      pad[i] ^= encrypted[full_block_length + i];
    for (unsigned int i = 0; i < c_asterisk_length; i++)
      out[full_block_length + i] = pad[i];
    // ^^p_asterisk
    for (unsigned int i = 0; i < c_asterisk_length; i++)
      checksum[i] ^= pad[i];
    checksum[c_asterisk_length] ^= 0x80;
  }
  xor_16(checksum, offset);
  xor_16(checksum, l_dollar);
  cipher(checksum, round_key);

  xor_16(checksum, &encrypted[encrypted_length]);
  unsigned char diff = 0;
  for (unsigned int i = 0; i < 16; i++)
    diff |= checksum[i];
  return (unsigned int) diff;
}

#ifdef __GNUC__
#undef USE_BUILTIN
#undef OCB_NONCEPARAM
#undef ocb_ntz
#undef ocb_ntz_round
#undef ocb_memcpy
#endif

#ifdef OCB_CONST_NONCE
#undef OCB_CONST_NONCE
#endif

#ifdef associated_data_length
#undef associated_data_length
#endif
